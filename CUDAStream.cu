#include "hip/hip_runtime.h"

// Copyright (c) 2015-16 Tom Deakin, Simon McIntosh-Smith,
// University of Bristol HPC
//
// For full license terms please see the LICENSE file distributed with this
// source code


#include "CUDAStream.h"

#define TBSIZE 1024

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

template <class T>
CUDAStream<T>::CUDAStream(const unsigned int ARRAY_SIZE, const int device_index)
{

  // The array size must be divisible by TBSIZE for kernel launches
  if (ARRAY_SIZE % TBSIZE != 0)
  {
    std::stringstream ss;
    ss << "Array size must be a multiple of " << TBSIZE;
    throw std::runtime_error(ss.str());
  }

  // Set device
  int count;
  hipGetDeviceCount(&count);
  check_error();
  if (device_index >= count)
    throw std::runtime_error("Invalid device index");
  hipSetDevice(device_index);
  check_error();

  // Print out device information
  std::cout << "Using CUDA device " << getDeviceName(device_index) << std::endl;
  std::cout << "Driver: " << getDeviceDriver(device_index) << std::endl;

  array_size = ARRAY_SIZE;

  // Check buffers fit on the device
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  if (props.totalGlobalMem < 3*ARRAY_SIZE*sizeof(T))
    throw std::runtime_error("Device does not have enough memory for all 3 buffers");

  // Create device buffers
  hipMalloc(&d_a, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_b, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_c, ARRAY_SIZE*sizeof(T));
  check_error();
}


template <class T>
CUDAStream<T>::~CUDAStream()
{
  hipFree(d_a);
  check_error();
  hipFree(d_b);
  check_error();
  hipFree(d_c);
  check_error();
}


template <typename T>
__global__ void init_kernel(T * a, T * b, T * c, T initA, T initB, T initC)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = initA;
  b[i] = initB;
  c[i] = initC;
}

template <class T>
void CUDAStream<T>::init_arrays(T initA, T initB, T initC)
{
  init_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, initA, initB, initC);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
void CUDAStream<T>::read_arrays(std::vector<T>& a, std::vector<T>& b, std::vector<T>& c)
{
  // Copy device memory to host
  hipMemcpy(a.data(), d_a, a.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(b.data(), d_b, b.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(c.data(), d_c, c.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
}


template <typename T>
__global__ void copy_kernel(const T * a, T * c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i];
}

template <class T>
void CUDAStream<T>::copy()
{
  copy_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void mul_kernel(T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  b[i] = scalar * c[i];
}

template <class T>
void CUDAStream<T>::mul()
{
  mul_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void add_kernel(const T * a, const T * b, T * c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

template <class T>
void CUDAStream<T>::add()
{
  add_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void triad_kernel(T * a, const T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = b[i] + scalar * c[i];
}

template <class T>
void CUDAStream<T>::triad()
{
  triad_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}


void listDevices(void)
{
  // Get number of devices
  int count;
  hipGetDeviceCount(&count);
  check_error();

  // Print device names
  if (count == 0)
  {
    std::cerr << "No devices found." << std::endl;
  }
  else
  {
    std::cout << std::endl;
    std::cout << "Devices:" << std::endl;
    for (int i = 0; i < count; i++)
    {
      std::cout << i << ": " << getDeviceName(i) << std::endl;
    }
    std::cout << std::endl;
  }
}


std::string getDeviceName(const int device)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  return std::string(props.name);
}


std::string getDeviceDriver(const int device)
{
  hipSetDevice(device);
  check_error();
  int driver;
  hipDriverGetVersion(&driver);
  check_error();
  return std::to_string(driver);
}

template class CUDAStream<float>;
template class CUDAStream<double>;
