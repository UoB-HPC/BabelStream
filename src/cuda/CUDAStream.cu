#include "hip/hip_runtime.h"

// Copyright (c) 2015-16 Tom Deakin, Simon McIntosh-Smith,
// University of Bristol HPC
//
// For full license terms please see the LICENSE file distributed with this
// source code


#include "CUDAStream.h"

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

template <class T>
CUDAStream<T>::CUDAStream(const int ARRAY_SIZE, const int device_index)
{

  // The array size must be divisible by TBSIZE for kernel launches
  if (ARRAY_SIZE % TBSIZE != 0)
  {
    std::stringstream ss;
    ss << "Array size must be a multiple of " << TBSIZE;
    throw std::runtime_error(ss.str());
  }

  // Set device
  int count;
  hipGetDeviceCount(&count);
  check_error();
  if (device_index >= count)
    throw std::runtime_error("Invalid device index");
  hipSetDevice(device_index);
  check_error();

  // Print out device information
  std::cout << "Using CUDA device " << getDeviceName(device_index) << std::endl;
  std::cout << "Driver: " << getDeviceDriver(device_index) << std::endl;

  array_size = ARRAY_SIZE;

  // Allocate the host array for partial sums for dot kernels
  sums = (T*)malloc(sizeof(T) * DOT_NUM_BLOCKS);

  size_t array_bytes = sizeof(T);
  array_bytes *= ARRAY_SIZE;
  size_t total_bytes = array_bytes * 3;

  // Check buffers fit on the device
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  if (props.totalGlobalMem < total_bytes)
    throw std::runtime_error("Device does not have enough memory for all 3 buffers");

  // Create device buffers
#if defined(MANAGED)
  hipMallocManaged(&d_a, array_bytes);
  check_error();
  hipMallocManaged(&d_b, array_bytes);
  check_error();
  hipMallocManaged(&d_c, array_bytes);
  check_error();
  hipMallocManaged(&d_sum, DOT_NUM_BLOCKS*sizeof(T));
  check_error();
#elif defined(PAGEFAULT)
  d_a = (T*)malloc(array_bytes);
  d_b = (T*)malloc(array_bytes);
  d_c = (T*)malloc(array_bytes);
  d_sum = (T*)malloc(sizeof(T)*DOT_NUM_BLOCKS);
#else
  hipMalloc(&d_a, array_bytes);
  check_error();
  hipMalloc(&d_b, array_bytes);
  check_error();
  hipMalloc(&d_c, array_bytes);
  check_error();
  hipMalloc(&d_sum, DOT_NUM_BLOCKS*sizeof(T));
  check_error();
#endif
}


template <class T>
CUDAStream<T>::~CUDAStream()
{
  free(sums);

#if defined(PAGEFAULT)
  free(d_a);
  free(d_b);
  free(d_c);
  free(d_sum);
#else
  hipFree(d_a);
  check_error();
  hipFree(d_b);
  check_error();
  hipFree(d_c);
  check_error();
  hipFree(d_sum);
  check_error();
#endif
}


template <typename T>
__global__ void init_kernel(T * a, T * b, T * c, T initA, T initB, T initC)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = initA;
  b[i] = initB;
  c[i] = initC;
}

template <class T>
void CUDAStream<T>::init_arrays(T initA, T initB, T initC)
{
  init_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c, initA, initB, initC);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
void CUDAStream<T>::read_arrays(std::vector<T>& a, std::vector<T>& b, std::vector<T>& c)
{
  // Copy device memory to host
#if defined(PAGEFAULT) || defined(MANAGED)
  hipDeviceSynchronize();
  for (int i = 0; i < array_size; i++)
  {
    a[i] = d_a[i];
    b[i] = d_b[i];
    c[i] = d_c[i];
  }
#else
  hipMemcpy(a.data(), d_a, a.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(b.data(), d_b, b.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(c.data(), d_c, c.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
#endif
}


template <typename T>
__global__ void copy_kernel(const T * a, T * c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i];
}

template <class T>
void CUDAStream<T>::copy()
{
  copy_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void mul_kernel(T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  b[i] = scalar * c[i];
}

template <class T>
void CUDAStream<T>::mul()
{
  mul_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void add_kernel(const T * a, const T * b, T * c)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  c[i] = a[i] + b[i];
}

template <class T>
void CUDAStream<T>::add()
{
  add_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void triad_kernel(T * a, const T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = b[i] + scalar * c[i];
}

template <class T>
void CUDAStream<T>::triad()
{
  triad_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void nstream_kernel(T * a, const T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] += b[i] + scalar * c[i];
}

template <class T>
void CUDAStream<T>::nstream()
{
  nstream_kernel<<<array_size/TBSIZE, TBSIZE>>>(d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
__global__ void dot_kernel(const T * a, const T * b, T * sum, int array_size)
{
  __shared__ T tb_sum[TBSIZE];

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  const size_t local_i = threadIdx.x;

  tb_sum[local_i] = 0.0;
  for (; i < array_size; i += blockDim.x*gridDim.x)
    tb_sum[local_i] += a[i] * b[i];

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
  {
    __syncthreads();
    if (local_i < offset)
    {
      tb_sum[local_i] += tb_sum[local_i+offset];
    }
  }

  if (local_i == 0)
    sum[blockIdx.x] = tb_sum[local_i];
}

template <class T>
T CUDAStream<T>::dot()
{
  dot_kernel<<<DOT_NUM_BLOCKS, TBSIZE>>>(d_a, d_b, d_sum, array_size);
  check_error();

#if defined(MANAGED) || defined(PAGEFAULT)
  hipDeviceSynchronize();
  check_error();
#else
  hipMemcpy(sums, d_sum, DOT_NUM_BLOCKS*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
#endif

  T sum = 0.0;
  for (int i = 0; i < DOT_NUM_BLOCKS; i++)
  {
#if defined(MANAGED) || defined(PAGEFAULT)
    sum += d_sum[i];
#else
    sum += sums[i];
#endif
  }

  return sum;
}

void listDevices(void)
{
  // Get number of devices
  int count;
  hipGetDeviceCount(&count);
  check_error();

  // Print device names
  if (count == 0)
  {
    std::cerr << "No devices found." << std::endl;
  }
  else
  {
    std::cout << std::endl;
    std::cout << "Devices:" << std::endl;
    for (int i = 0; i < count; i++)
    {
      std::cout << i << ": " << getDeviceName(i) << std::endl;
    }
    std::cout << std::endl;
  }
}


std::string getDeviceName(const int device)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  return std::string(props.name);
}


std::string getDeviceDriver(const int device)
{
  hipSetDevice(device);
  check_error();
  int driver;
  hipDriverGetVersion(&driver);
  check_error();
  return std::to_string(driver);
}

template class CUDAStream<float>;
template class CUDAStream<double>;
