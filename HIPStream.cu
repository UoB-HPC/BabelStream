#include "hip/hip_runtime.h"
// Copyright (c) 2015-16 Tom Deakin, Simon McIntosh-Smith,
// University of Bristol HPC
//
// For full license terms please see the LICENSE file distributed with this
// source code


#include "HIPStream.h"
#include "hip/hip_runtime.h"

#define TBSIZE 1024

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

template <class T>
HIPStream<T>::HIPStream(const unsigned int ARRAY_SIZE, const int device_index)
{

  // The array size must be divisible by TBSIZE for kernel launches
  if (ARRAY_SIZE % TBSIZE != 0)
  {
    std::stringstream ss;
    ss << "Array size must be a multiple of " << TBSIZE;
    throw std::runtime_error(ss.str());
  }

  // Set device
  int count;
  hipGetDeviceCount(&count);
  check_error();
  if (device_index >= count)
    throw std::runtime_error("Invalid device index");
  hipSetDevice(device_index);
  check_error();

  // Print out device information
  std::cout << "Using HIP device " << getDeviceName(device_index) << std::endl;
  std::cout << "Driver: " << getDeviceDriver(device_index) << std::endl;

  array_size = ARRAY_SIZE;

  // Check buffers fit on the device
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  if (props.totalGlobalMem < 3*ARRAY_SIZE*sizeof(T))
    throw std::runtime_error("Device does not have enough memory for all 3 buffers");

  // Create device buffers
  hipMalloc(&d_a, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_b, ARRAY_SIZE*sizeof(T));
  check_error();
  hipMalloc(&d_c, ARRAY_SIZE*sizeof(T));
  check_error();
}


template <class T>
HIPStream<T>::~HIPStream()
{
  hipFree(d_a);
  check_error();
  hipFree(d_b);
  check_error();
  hipFree(d_c);
  check_error();
}

template <typename T>
__global__ void init_kernel(hipLaunchParm lp, T * a, T * b, T * c, T initA, T initB, T initC)
{
  const int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
  a[i] = initA;
  b[i] = initB;
  c[i] = initC;
}

template <class T>
void HIPStream<T>::init_arrays(T initA, T initB, T initC)
{
  hipLaunchKernel(HIP_KERNEL_NAME(init_kernel), dim3(array_size/TBSIZE), dim3(TBSIZE), 0, 0, d_a, d_b, d_c, initA, initB, initC);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <class T>
void HIPStream<T>::read_arrays(std::vector<T>& a, std::vector<T>& b, std::vector<T>& c)
{
  // Copy device memory to host
  hipMemcpy(a.data(), d_a, a.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(b.data(), d_b, b.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
  hipMemcpy(c.data(), d_c, c.size()*sizeof(T), hipMemcpyDeviceToHost);
  check_error();
}


template <typename T>
__global__ void copy_kernel(hipLaunchParm lp, const T * a, T * c)
{
  const int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
  c[i] = a[i];
}

template <class T>
void HIPStream<T>::copy()
{
  hipLaunchKernel(HIP_KERNEL_NAME(copy_kernel), dim3(array_size/TBSIZE), dim3(TBSIZE), 0, 0, d_a, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void mul_kernel(hipLaunchParm lp, T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
  b[i] = scalar * c[i];
}

template <class T>
void HIPStream<T>::mul()
{
  hipLaunchKernel(HIP_KERNEL_NAME(mul_kernel), dim3(array_size/TBSIZE), dim3(TBSIZE), 0, 0, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void add_kernel(hipLaunchParm lp, const T * a, const T * b, T * c)
{
  const int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
  c[i] = a[i] + b[i];
}

template <class T>
void HIPStream<T>::add()
{
  hipLaunchKernel(HIP_KERNEL_NAME(add_kernel), dim3(array_size/TBSIZE), dim3(TBSIZE), 0, 0, d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}

template <typename T>
__global__ void triad_kernel(hipLaunchParm lp, T * a, const T * b, const T * c)
{
  const T scalar = startScalar;
  const int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
  a[i] = b[i] + scalar * c[i];
}

template <class T>
void HIPStream<T>::triad()
{
  hipLaunchKernel(HIP_KERNEL_NAME(triad_kernel), dim3(array_size/TBSIZE), dim3(TBSIZE), 0, 0, d_a, d_b, d_c);
  check_error();
  hipDeviceSynchronize();
  check_error();
}


void listDevices(void)
{
  // Get number of devices
  int count;
  hipGetDeviceCount(&count);
  check_error();

  // Print device names
  if (count == 0)
  {
    std::cerr << "No devices found." << std::endl;
  }
  else
  {
    std::cout << std::endl;
    std::cout << "Devices:" << std::endl;
    for (int i = 0; i < count; i++)
    {
      std::cout << i << ": " << getDeviceName(i) << std::endl;
    }
    std::cout << std::endl;
  }
}


std::string getDeviceName(const int device)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  return std::string(props.name);
}


std::string getDeviceDriver(const int device)
{
  hipSetDevice(device);
  check_error();
  int driver;
  hipDriverGetVersion(&driver);
  check_error();
  return std::to_string(driver);
}

template class HIPStream<float>;
template class HIPStream<double>;
